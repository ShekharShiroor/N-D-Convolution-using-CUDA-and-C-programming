#include "hip/hip_runtime.h"
#include <wb.h>

#define MASK_WIDTH 5
#define O_TILE_WIDTH 16
#define clamp(x) (min(max((x), 0.0), 1.0))
#define BLOCK_WIDTH (O_TILE_WIDTH + MASK_WIDTH - 1)
#define mask_radius (MASK_WIDTH / 2)

//@@ INSERT CODE HERE 
//implement the tiled 2D convolution kernel with adjustments for channels
//use shared memory to reduce the number of global accesses, handle the boundary conditions when loading input list elements into the shared memory
//clamp your output values
__global__ void convolution_2D_kernel(float *P, const float* __restrict__ N, int height, int width, int channels, const float* __restrict__ M)
{
	__shared__ float Ns[BLOCK_WIDTH][BLOCK_WIDTH];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row_o = blockIdx.y*O_TILE_WIDTH + ty;
	int col_o = blockIdx.x*O_TILE_WIDTH + tx;
	int row_i = row_o - mask_radius;
	int col_i = col_o - mask_radius;

		if ((row_i >= 0) && (row_i < height) && (col_i >= 0) && (col_i < width))
		{
			Ns[ty][tx] = N[(row_i * width + col_i) * channels + blockIdx.z];
		}
		else {
			Ns[ty][tx] = 0.0f;
		}

		__syncthreads();

		float output = 0.0f;
		if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH)
		{
			for (int i = 0; i < MASK_WIDTH; i++) {
				for (int j = 0; j < MASK_WIDTH; j++) {
					output += M[i * MASK_WIDTH + j] * Ns[i + ty][j + tx];
				}
			}


			if (row_o < height && col_o < width)
				P[(row_o*width + col_o) * channels + blockIdx.z] = clamp(output);
		}

}

int main(int argc, char *argv[]) {
	wbArg_t arg;
	int maskRows;
	int maskColumns;
	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	char *inputMaskFile;
	wbImage_t inputImage;
	wbImage_t outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *hostMaskData;
	float *deviceInputImageData;
	float *deviceOutputImageData;
	float *deviceMaskData;

	arg = wbArg_read(argc, argv); /* parse the input arguments */

	inputImageFile = wbArg_getInputFile(arg, 0);
	inputMaskFile = wbArg_getInputFile(arg, 1);

	inputImage = wbImport(inputImageFile);
	hostMaskData = (float *)wbImport(inputMaskFile, &maskRows, &maskColumns);

	assert(maskRows == MASK_WIDTH);    /* mask height is fixed to 5 */
	assert(maskColumns == MASK_WIDTH); /* mask width is fixed to 5 */

	imageWidth = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage);

	outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

	hostInputImageData = wbImage_getData(inputImage);
	hostOutputImageData = wbImage_getData(outputImage);

	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	wbTime_start(GPU, "Doing GPU memory allocation");
	//@@ INSERT CODE HERE
	//allocate device memory
	int sizeMask = sizeof(float) * maskColumns * maskRows;
	int sizeInputImage = sizeof(float) * imageWidth * imageHeight * imageChannels;

	hipMalloc((void**)&deviceMaskData, sizeMask);
	hipMalloc((void**)&deviceInputImageData, sizeInputImage);
	hipMalloc((void**)&deviceOutputImageData, sizeInputImage);



	wbTime_stop(GPU, "Doing GPU memory allocation");

	wbTime_start(Copy, "Copying data to the GPU");
	//@@ INSERT CODE HERE
	//copy host memory to device
	hipMemcpy(deviceMaskData, hostMaskData, sizeMask, hipMemcpyHostToDevice);
	hipMemcpy(deviceInputImageData, hostInputImageData, sizeInputImage, hipMemcpyHostToDevice);


	wbTime_stop(Copy, "Copying data to the GPU");

	wbTime_start(Compute, "Doing the computation on the GPU");
	//@@ INSERT CODE HERE
	//initialize thread block and kernel grid dimensions
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
	dim3 dimGrid(((imageWidth - 1) / O_TILE_WIDTH) + 1, ((imageHeight - 1) / O_TILE_WIDTH) + 1, imageChannels);

	//invoke CUDA kernel	
	convolution_2D_kernel << < dimGrid, dimBlock >> > (deviceOutputImageData, deviceInputImageData, imageHeight, imageWidth, imageChannels, deviceMaskData);

	wbTime_stop(Compute, "Doing the computation on the GPU");

	wbTime_start(Copy, "Copying data from the GPU");
	//@@ INSERT CODE HERE
	//copy results from device to host	
	hipMemcpy(hostOutputImageData, deviceOutputImageData, sizeInputImage, hipMemcpyDeviceToHost);


	wbTime_stop(Copy, "Copying data from the GPU");

	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	wbSolution(arg, outputImage);

	//@@ INSERT CODE HERE
	//deallocate device memory	
	hipFree(deviceMaskData);
	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);

	free(hostMaskData);
	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

	return 0;
}
